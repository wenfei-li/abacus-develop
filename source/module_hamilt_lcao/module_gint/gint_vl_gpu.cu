#include <omp.h>

#include "kernels/cuda/cuda_tools.cuh"
#include "module_base/ylm.h"
#include "gint_tools.h"
#include "gint_vl_gpu.h"
#include "kernels/cuda/gint_vl.cuh"

namespace GintKernel
{

/**
 * Computes the gamma component of the VL (Vlocal) integral on the GPU.
 *
 * @param hRGint Pointer to the HContainer<double> object to store the computed
 * integrals.
 * @param lgd Dimension information for the computation results.
 * @param max_atom The maximum number of neighboring atoms for a grid point.
 * @param vfactor Related to volume. The scaling factor for the Vlocal
 * integrals.
 * @param vlocal Pointer to the Vlocal array.
 * @param ylmcoef_now Pointer to the Ylm coefficients array.
 * @param nczp The number of grid layers in the C direction.
 * @param nbxx The total number of grid points.
 * @param dr The grid spacing.
 * @param rcut Pointer to the cutoff radius array.
 * @param gridt The Grid_Technique object containing grid information.
 * @param ucell The UnitCell object containing unit cell information.
 *
 * @note The grid integration on the GPU is mainly divided into the following
 * steps:
 * 1. Use the CPU to divide the grid integration into subtasks.
 * 2. Copy the subtask information to the GPU.
 * 3. Calculate the matrix elements on the GPU.
 * 4. Perform matrix multiplication on the GPU.
 * 5. Copy the results back to the host.
 */
void gint_gamma_vl_gpu(hamilt::HContainer<double>* hRGint,
                       const double* vlocal,
                       const double* ylmcoef_now,
                       const double dr,
                       const double* rcut,
                       const Grid_Technique& gridt,
                       const UnitCell& ucell)
{


    int dev_id = base_device::information::set_device_by_rank();
    checkCuda(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
    const int nbzp = gridt.nbzp;
    const int num_streams = gridt.nstreams;
    const int lgd = gridt.lgd;
    const int max_atom = gridt.max_atom;
    const int max_atom_per_bcell = max_atom * gridt.bxyz;
    const int max_atom_per_z = max_atom_per_bcell * nbzp;
    const int max_phi_per_z = max_atom_per_z * ucell.nwmax;
    const int max_atompair_per_z = max_atom * max_atom * nbzp;
    const double vfactor = ucell.omega / gridt.ncxyz;
    const int nczp = nbzp * gridt.bz;
    const int nbxx = gridt.nbxx;  // total number of grid points
    std::vector<hipStream_t> streams(num_streams);

    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamCreate(&streams[i]));
    }

    std::vector<Cuda_Mem_Wrapper<double>> grid_vlocal_g(ucell.nat * ucell.nat);
    for (int iat1 = 0; iat1 < ucell.nat; iat1++)
    {
        for (int iat2 = 0; iat2 < ucell.nat; iat2++)
        {
            const int it1 = ucell.iat2it[iat1];
            const int lo1 = gridt.trace_lo[ucell.itiaiw2iwt(it1,
                                                        ucell.iat2ia[iat1],
                                                        0)];

            const int it2 = ucell.iat2it[iat2];
            const int lo2 = gridt.trace_lo[ucell.itiaiw2iwt(it2,
                                                        ucell.iat2ia[iat2],
                                                        0)];

            if (lo1 <= lo2)
            {
                const hamilt::AtomPair<double>* tmp_ap
                    = hRGint->find_pair(iat1, iat2);
                if (tmp_ap == nullptr)
                {
                    continue;
                }
                const int atom_pair_nw
                    = ucell.atoms[it1].nw * ucell.atoms[it2].nw;
                grid_vlocal_g[iat1 * ucell.nat + iat2] = 
                    Cuda_Mem_Wrapper<double>(atom_pair_nw, 1, false);
                grid_vlocal_g[iat1 * ucell.nat + iat2].memset_device_sync();
            }
        }
    }

    Cuda_Mem_Wrapper<double> psi_input_double(5 * max_atom_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> psi_input_int(2 * max_atom_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> atom_num_per_bcell(nbzp, num_streams, true);
    Cuda_Mem_Wrapper<int> start_idx_per_bcell(nbzp, num_streams, true);

    Cuda_Mem_Wrapper<double> psi(max_phi_per_z, num_streams, false);
    Cuda_Mem_Wrapper<double> psi_vldr3(max_phi_per_z, num_streams, false);

    Cuda_Mem_Wrapper<int> gemm_m(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_n(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_k(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_lda(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldb(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldc(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_A(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_B(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_C(max_atompair_per_z, num_streams, true);

#pragma omp parallel for num_threads(num_streams) collapse(2)
    for (int i = 0; i < gridt.nbx; i++)
    {
        for (int j = 0; j < gridt.nby; j++)
        {
            // 20240620 Note that it must be set again here because 
            // cuda's device is not safe in a multi-threaded environment.

            checkCuda(hipSetDevice(dev_id));
            const int sid = omp_get_thread_num();
            checkCuda(hipStreamSynchronize(streams[sid]));

            int max_m = 0;
            int max_n = 0;
            int atom_pair_num = 0;
            int atom_per_z = 0;
            const int grid_index_ij = i * gridt.nby * gridt.nbzp + j * gridt.nbzp;
            std::vector<bool> gpu_matrix_calc_flag(max_atom * nbzp, false);
            gtask_vlocal(gridt,
                         rcut,
                         ucell,
                         gpu_matrix_calc_flag,
                         grid_index_ij,
                         max_atom,
                         nczp,
                         vfactor,
                         vlocal,
                         psi_input_double.get_host_pointer(sid),
                         psi_input_int.get_host_pointer(sid),
                         atom_num_per_bcell.get_host_pointer(sid),
                         start_idx_per_bcell.get_host_pointer(sid),
                         atom_per_z);
        
            alloc_mult_vlocal(gridt,
                                ucell,
                                gpu_matrix_calc_flag,
                                grid_index_ij,
                                max_atom,
                                psi.get_device_pointer(sid),
                                psi_vldr3.get_device_pointer(sid),
                                grid_vlocal_g,
                                gemm_m.get_host_pointer(sid),
                                gemm_n.get_host_pointer(sid),
                                gemm_k.get_host_pointer(sid),
                                gemm_lda.get_host_pointer(sid),
                                gemm_ldb.get_host_pointer(sid),
                                gemm_ldc.get_host_pointer(sid),
                                gemm_A.get_host_pointer(sid),
                                gemm_B.get_host_pointer(sid),
                                gemm_C.get_host_pointer(sid),
                                atom_pair_num,
                                max_m,
                                max_n);

            psi_input_double.copy_host_to_device_async(streams[sid], sid, 5 * atom_per_z);
            psi_input_int.copy_host_to_device_async(streams[sid], sid, 2 * atom_per_z);
            atom_num_per_bcell.copy_host_to_device_async(streams[sid], sid);
            start_idx_per_bcell.copy_host_to_device_async(streams[sid], sid);
            gemm_m.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_n.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_k.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_lda.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldb.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldc.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_A.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_B.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_C.copy_host_to_device_async(streams[sid], sid, atom_pair_num);

            psi.memset_device_async(streams[sid], sid, 0);
            psi_vldr3.memset_device_async(streams[sid], sid, 0);

            dim3 grid_psi(nbzp, 8);
            dim3 block_psi(64);
            get_psi_and_vldr3<<<grid_psi,
                                block_psi,
                                0,
                                streams[sid]>>>(
                gridt.ylmcoef_g,
                dr,
                gridt.bxyz,
                ucell.nwmax,
                psi_input_double.get_device_pointer(sid),
                psi_input_int.get_device_pointer(sid),
                atom_num_per_bcell.get_device_pointer(sid),
                start_idx_per_bcell.get_device_pointer(sid),
                gridt.atom_nwl_g,
                gridt.atom_new_g,
                gridt.atom_ylm_g,
                gridt.atom_nw_g,
                gridt.nr_max,
                gridt.psi_u_g,
                psi.get_device_pointer(sid),
                psi_vldr3.get_device_pointer(sid));
            checkCudaLastError();
            
            gridt.fastest_matrix_mul(max_m,
                                     max_n,
                                     gemm_m.get_device_pointer(sid),
                                     gemm_n.get_device_pointer(sid),
                                     gemm_k.get_device_pointer(sid),
                                     gemm_A.get_device_pointer(sid),
                                     gemm_lda.get_device_pointer(sid),
                                     gemm_B.get_device_pointer(sid),
                                     gemm_ldb.get_device_pointer(sid),
                                     gemm_C.get_device_pointer(sid),
                                     gemm_ldc.get_device_pointer(sid),
                                     atom_pair_num,
                                     streams[sid],
                                     nullptr);
            checkCudaLastError();
        }
    }

    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamSynchronize(streams[i]));
    }
    {
        int iter_num = 0;
        for (int iat1 = 0; iat1 < ucell.nat; iat1++)
        {
            for (int iat2 = 0; iat2 < ucell.nat; iat2++)
            {
                const int sid = iter_num % num_streams;
                const int it1 = ucell.iat2it[iat1];
                const int lo1 = gridt.trace_lo[ucell.itiaiw2iwt(it1,
                                                          ucell.iat2ia[iat1],
                                                          0)];

                const int it2 = ucell.iat2it[iat2];
                const int lo2 = gridt.trace_lo[ucell.itiaiw2iwt(it2,
                                                          ucell.iat2ia[iat2],
                                                          0)];
                if (lo1 <= lo2)
                {
                    const int atom_pair_nw
                        = ucell.atoms[it1].nw * ucell.atoms[it2].nw;
                    hamilt::AtomPair<double>* tmp_ap
                        = hRGint->find_pair(iat1, iat2);
                    if (tmp_ap == nullptr)
                    {
                        continue;
                    }
                    checkCuda(hipMemcpyAsync(
                        tmp_ap->get_pointer(0),
                        grid_vlocal_g[iat1 * ucell.nat + iat2].get_device_pointer(),
                        atom_pair_nw * sizeof(double),
                        hipMemcpyDeviceToHost,
                        streams[sid]));
                    iter_num++;
                }
            }
        }
    }
    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamSynchronize(streams[i]));
    }
    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamDestroy(streams[i]));
    }
}

} // namespace GintKernel