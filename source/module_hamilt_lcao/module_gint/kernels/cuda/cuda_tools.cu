#include <iostream>
#include <cstring>

#include "cuda_tools.cuh"

hipError_t check(hipError_t result, const char *const func, const char *const file, const int line)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error at %s:%d code=%s \"%s\" \n", file, line, hipGetErrorString(result), func);
        exit(EXIT_FAILURE);
    }
    return result;
}
hipError_t __checkCudaLastError(const char *file, const int line)
{
    hipError_t result = hipGetLastError();
    if (result != hipSuccess)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError():%s\n", file, line, hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void dump_cuda_array_to_file(double* cuda_array,
                             int width,
                             int hight,
                             const std::string& filename)
{
    double* h_data = new double[width * hight];
    hipMemcpy(h_data,
               cuda_array,
               width * hight * sizeof(double),
               hipMemcpyDeviceToHost);

    std::ofstream outFile(filename);
    if (!outFile.is_open())
    {
        std::cerr << "Failed to open file for writing." << std::endl;
    }
    for (int j = 0; j < hight; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            outFile << "hight" << j << "   width:" << i << "   "
                    << h_data[j * width + i] << std::endl;
        }
    }
    outFile.close();
    delete[] h_data;
}

template <typename T>
Cuda_Mem_Wrapper<T>::Cuda_Mem_Wrapper()
{
    this->device_pointer = nullptr;
    this->host_pointer = nullptr;
    this->one_stream_size = 0;
    this->one_stream_size_aligned = 0;
    this->stream_number = 1;
    this->total_size_aligned = 0;
}

template <typename T>
Cuda_Mem_Wrapper<T>::Cuda_Mem_Wrapper(int one_stream_size_in,
                                      int one_stream_size_aligned_in,
                                      int stream_number_in,
                                      bool malloc_host_in)
{
    this->stream_number = stream_number_in;
    this->one_stream_size = one_stream_size_in;
    this->one_stream_size_aligned = one_stream_size_aligned_in;
    this->total_size_aligned
        = this->one_stream_size_aligned * this->stream_number;

    checkCuda(hipMalloc((void**)&this->device_pointer,
                         this->total_size_aligned * sizeof(T)));
    checkCuda(hipMemset(this->device_pointer,
                         0,
                         this->total_size_aligned * sizeof(T)));
    this->host_pointer = nullptr;

    if (malloc_host_in)
    {
        checkCuda(hipHostMalloc((void**)&this->host_pointer,
                                 this->total_size_aligned * sizeof(T)));
        memset(this->host_pointer, 0, this->total_size_aligned * sizeof(T));
    }
}

template <typename T>
Cuda_Mem_Wrapper<T>::Cuda_Mem_Wrapper(int one_stream_size_in,
                                      int stream_number_in,
                                      bool malloc_host_in)
    : Cuda_Mem_Wrapper(one_stream_size_in,
                       one_stream_size_in,
                       stream_number_in,
                       malloc_host_in)
{
}

template <typename T>
Cuda_Mem_Wrapper<T>::Cuda_Mem_Wrapper(Cuda_Mem_Wrapper&& other) noexcept
{
    this->device_pointer = other.device_pointer;
    this->host_pointer = other.host_pointer;
    this->one_stream_size = other.one_stream_size;
    this->one_stream_size_aligned = other.one_stream_size_aligned;
    this->stream_number = other.stream_number;
    this->total_size_aligned = other.total_size_aligned;

    other.device_pointer = nullptr;
    other.host_pointer = nullptr;
    other.one_stream_size = 0;
    other.one_stream_size_aligned = 0;
    other.stream_number = 0;
    other.total_size_aligned = 0;
}

template <typename T>
Cuda_Mem_Wrapper<T>& Cuda_Mem_Wrapper<T>::operator=(Cuda_Mem_Wrapper&& other) noexcept
{
    if (this != &other)
    {
        this->free_all();
        this->device_pointer = other.device_pointer;
        this->host_pointer = other.host_pointer;
        this->one_stream_size = other.one_stream_size;
        this->one_stream_size_aligned = other.one_stream_size_aligned;
        this->stream_number = other.stream_number;
        this->total_size_aligned = other.total_size_aligned;

        other.device_pointer = nullptr;
        other.host_pointer = nullptr;
        other.one_stream_size = 0;
        other.one_stream_size_aligned = 0;
        other.stream_number = 0;
        other.total_size_aligned = 0;
    }
    return *this;
}

template <typename T>
void Cuda_Mem_Wrapper<T>::free_all()
{
    checkCuda(hipFree(this->device_pointer));
    if (this->host_pointer != nullptr)
    {
        checkCuda(hipHostFree(this->host_pointer));
    }
}

template <typename T>
Cuda_Mem_Wrapper<T>::~Cuda_Mem_Wrapper()
{
    this->free_all();
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::copy_host_to_device_sync(const int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy host to device"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpy(
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyHostToDevice));
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::copy_host_to_device_async(const hipStream_t stream,
                                                    const int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy host to device"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpyAsync(
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyHostToDevice,
        stream));
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::copy_host_to_device_async(const hipStream_t stream,
                                                    const int stream_id,
                                                    const int size)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy host to device"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpyAsync(
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        size * sizeof(T),
        hipMemcpyHostToDevice,
        stream));
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::copy_device_to_host_sync(const int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy device to host"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpy(
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyDeviceToHost));
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::copy_device_to_host_async(const hipStream_t stream,
                                                    const int stream_id)
{
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy device to host"
                  << std::endl;
        exit(1);
    }
    checkCuda(hipMemcpyAsync(
        this->host_pointer + stream_id * this->one_stream_size_aligned,
        this->device_pointer + stream_id * this->one_stream_size_aligned,
        this->one_stream_size * sizeof(T),
        hipMemcpyDeviceToHost,
        stream));
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::memset_device_sync(const int stream_id, const int value)
{
    checkCuda(hipMemset(this->device_pointer
                             + stream_id * this->one_stream_size_aligned,
                         value,
                         this->one_stream_size * sizeof(T)));
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::memset_device_async(const hipStream_t stream,
                                              const int stream_id,
                                              const int value)
{
    checkCuda(hipMemsetAsync(this->device_pointer
                                  + stream_id * this->one_stream_size_aligned,
                              value,
                              this->one_stream_size * sizeof(T),
                              stream));
}

template <typename T>
inline void Cuda_Mem_Wrapper<T>::memset_host(const int stream_id, const int value)
{
    memset(this->host_pointer + stream_id * this->one_stream_size_aligned,
           value,
           this->one_stream_size * sizeof(T));
}

template <typename T>
inline T* Cuda_Mem_Wrapper<T>::get_device_pointer(const int stream_id)
{
    return this->device_pointer + stream_id * this->one_stream_size_aligned;
}

template <typename T>
inline T* Cuda_Mem_Wrapper<T>::get_host_pointer(const int stream_id)
{
    return this->host_pointer + stream_id * this->one_stream_size_aligned;
}
template class Cuda_Mem_Wrapper<double>;
template class Cuda_Mem_Wrapper<double*>;
template class Cuda_Mem_Wrapper<int>;
