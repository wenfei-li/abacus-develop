#include "hip/hip_runtime.h"
#include "module_hamilt_pw/hamilt_pwdft/kernels/veff_op.h"

#include <complex>
#include <thrust/complex.h>

#include <hip/hip_runtime.h>
#include <base/macros/macros.h>

namespace hamilt{

#define THREADS_PER_BLOCK 256

template <typename FPTYPE>
__global__ void veff_pw(
    const int size,
    thrust::complex<FPTYPE>* out,
    const FPTYPE* in)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) {return;}
    out[idx] *= in[idx];
}

template <typename FPTYPE>
__global__ void veff_pw(
    const int size,
    thrust::complex<FPTYPE>* out,
    thrust::complex<FPTYPE>* out1,
    const FPTYPE* in)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) {return;}
    thrust::complex<FPTYPE> sup =
        out[idx] * (in[0 * size + idx] + in[3 * size + idx])
            + out1[idx] * (in[1 * size + idx] - thrust::complex<FPTYPE>(0.0, 1.0) * in[2 * size + idx]);
    thrust::complex<FPTYPE> sdown =
        out1[idx] * (in[0 * size + idx] - in[3 * size + idx])
            + out[idx] * (in[1 * size + idx] + thrust::complex<FPTYPE>(0.0, 1.0) * in[2 * size + idx]);
    out[idx] = sup;
    out1[idx] = sdown;
}

template <typename FPTYPE>
void veff_pw_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev,
                                                             const int& size,
                                                             std::complex<FPTYPE>* out,
                                                             const FPTYPE* in)
{
    const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipLaunchKernelGGL(HIP_KERNEL_NAME(veff_pw<FPTYPE>), dim3(block), dim3(THREADS_PER_BLOCK), 0, 0,
        size, // control params
        reinterpret_cast<thrust::complex<FPTYPE>*>(out), // array of data
        in); // array of data

    hipCheckOnDebug();
}

template <typename FPTYPE>
void veff_pw_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev,
                                                             const int& size,
                                                             std::complex<FPTYPE>* out,
                                                             std::complex<FPTYPE>* out1,
                                                             const FPTYPE** in)
{
    const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipLaunchKernelGGL(HIP_KERNEL_NAME(veff_pw<FPTYPE>), dim3(block), dim3(THREADS_PER_BLOCK), 0, 0,
        size, // control params
        reinterpret_cast<thrust::complex<FPTYPE>*>(out), // array of data
        reinterpret_cast<thrust::complex<FPTYPE>*>(out1), // array of data
        in[0]); // array of data

    hipCheckOnDebug();
}

template struct veff_pw_op<float, base_device::DEVICE_GPU>;
template struct veff_pw_op<double, base_device::DEVICE_GPU>;

}  // namespace hamilt